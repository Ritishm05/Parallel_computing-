
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1024

__global__ void ans(int *inp, int *out) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) {
        int ans = 0;
        for (int i = 0; i < N; i++) {
            ans += inp[i];
        }
        *out = ans;
    }
}

int main() {
    int *h_inp, *h_out;
    int *d_inp, *d_out;

    size_t size = N * sizeof(int);
    h_inp = (int *)malloc(size);
    h_out = (int *)malloc(sizeof(int));

    for (int i = 0; i < N; i++) {
        h_inp[i] = i + 1;
    }

    hipMalloc((void **)&d_inp, size);
    hipMalloc((void **)&d_out, sizeof(int));

    hipMemcpy(d_inp, h_inp, size, hipMemcpyHostToDevice);
    hipMemset(d_out, 0, sizeof(int));
    ans<<<1, 32>>>(d_inp, d_out);
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

    printf("ans of first %d integers is: %d\n", N, *h_out);

    free(h_inp);
    free(h_out);
    hipFree(d_inp);
    hipFree(d_out);

    return 0;
}
